#include "hip/hip_runtime.h"
#include <math.h>
#include <hip/hip_runtime.h>
#include "LSTM.cuh"

template<typename T>
__global__ void mat_tanh_kernel(const T* A, T* C, int p, int q) {

        //Addition by reading from Global Memory

printf("kernel call tanh: %d, blockdim.x = %d, blockdim.y = %d\n", threadIdx.x, blockDim.x, blockDim.y);
	int A_index = (blockDim.y)*(blockIdx.y)*q + (threadIdx.y)*q + (blockDim.x)*(blockIdx.x) + threadIdx.x;
	int C_index = (blockDim.y)*(blockIdx.y)*q + (threadIdx.y)*q + (blockDim.x)*(blockIdx.x) + threadIdx.x;

        C[C_index] = tanhf(A[A_index]);

}

__host__ void LSTM::mat_tanh_gpu(const float *A, float *C, int p, int q){

    //max threads per block needs to taken care while using dim3. Else kernel wont launch.
    dim3 dimBlock(m_block_dim, m_block_dim, 1);
	dim3 dimGrid(((p+dimBlock.x-1)/dimBlock.x), ((q+dimBlock.y-1)/dimBlock.y), 1); // There may be some incomplete(not block_dimXblock_dim) blocks at the edges
	//kernel Launch
	mat_tanh_kernel<float><<<dimGrid, dimBlock>>>(A, C, p, q);
	     hipDeviceSynchronize();

}
