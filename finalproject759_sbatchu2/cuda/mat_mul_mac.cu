#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "LSTM.cuh"
using namespace std;

//Tiling is done across shared memories present in all SMs. 
//Each shared memory will get m_block_dim X m_block_dim amount of data each of A, B and compute C tile corresponding to them.
//standard (m,n,k) = (p,r,q) : k/q dimension is accumulating dimension for each output tile.
template<typename T>
__global__ void mat_mul_mac_kernel(const T* A, const T* B, T*C, unsigned int p, unsigned int q, unsigned int r, int accum_mode){

//Dynamic shared memory declared
extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
T *shared_mem = reinterpret_cast<T *>(my_smem);

//assigning shared memory to tiled A,B,C pointers
T* tileA = (T*)shared_mem;
T* tileB = (T*)&shared_mem[blockDim.x*blockDim.y];
//T* tileC = (T*)&s[2*blockDim.x*blockDim.y];

printf("kernel call mat_mul_mac : %d, blockdim.x = %d, blockdim.y = %d\n", threadIdx.x, blockDim.x, blockDim.y);
T dot_sum = 0;
for(unsigned int i=0; i<((q+blockDim.x-1)/blockDim.x); i++)
{
	unsigned long int A_index = blockIdx.y*q*blockDim.y+i*blockDim.x+threadIdx.x+threadIdx.y*q;
	unsigned int A_index_x = i*blockDim.x+threadIdx.x;
	unsigned int A_index_y = blockIdx.y*blockDim.y + threadIdx.y;
	if((A_index < p*q) && (A_index_x < q) && (A_index_y < p))
	    tileA[threadIdx.x+blockDim.x*threadIdx.y] = A[A_index];
    else 
	    tileA[threadIdx.x+blockDim.x*threadIdx.y] = 0;

	//printf("tile A thread x : %d, thread y : %d, value : %f\n", threadIdx.x, threadIdx.y,  tileA[threadIdx.x+blockDim.x*threadIdx.y]);
	
	unsigned long int B_index = blockIdx.x*blockDim.x+i*r*blockDim.y+threadIdx.x+threadIdx.y*r;
    unsigned int B_index_x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int B_index_y = i*blockDim.y + threadIdx.y;
	if((B_index < q*r) && (B_index_x < r) && (B_index_y < q))
	    tileB[threadIdx.x+blockDim.x*threadIdx.y] = B[B_index];
    else
	    tileB[threadIdx.x+blockDim.x*threadIdx.y] = 0;
    //printf("tile B thread x : %d, thread y : %d, value : %f\n", threadIdx.x, threadIdx.y,  tileB[threadIdx.x+blockDim.x*threadIdx.y]);

	//sync to make sure data is reached to shared memory
	__syncthreads();

   

	for(int k =0; k<blockDim.x; k++)
	{
         dot_sum += tileA[k+threadIdx.y*blockDim.x]*tileB[threadIdx.x + k*blockDim.x];
	}
	//printf("tile C thread x : %d, thread y : %d, value : %f\n", threadIdx.x, threadIdx.y, dot_sum);
	__syncthreads();
}
    
	unsigned long int C_index = blockIdx.y*blockDim.y*r+threadIdx.y*r+blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int C_index_x = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int C_index_y = blockIdx.y*blockDim.y+threadIdx.y;
    if((C_index < p*r) && (C_index_x < r) && (C_index_y < p)) {
	        if(accum_mode == 1) {
		   C[C_index] += dot_sum;
		} else {
		   C[C_index] = dot_sum;
		}
	} 

}


void LSTM::mat_mac_gpu(const float *A, const float *B, float *C, int p, int q, int r){

    printf("gpu about to call kernel1 mat_mul_mac , block_dim : %d, p : %d, q : %d, r : %d\n", m_block_dim, p, q, r);
    //max threads per block needs to taken care while using dim3. Else kernel wont launch.
    dim3 dimBlock(m_block_dim, m_block_dim, 1);
	dim3 dimGrid(((p+dimBlock.x-1)/dimBlock.x), ((r+dimBlock.y-1)/dimBlock.y), 1); // There may be some incomplete(not block_dimXblock_dim) blocks at the edges
	//kernel Launch
	mat_mul_mac_kernel<float><<<dimGrid, dimBlock, 2*(m_block_dim*m_block_dim)*sizeof(float)>>>(A, B, C, p, q, r, 1);
    hipDeviceSynchronize();
    printf("gpu about to call kernel2 mat_mul_mac , block_dim : %d, p : %d, q : %d, r : %d\n", m_block_dim, p, q, r);
}

