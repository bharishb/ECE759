#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "LSTM.cuh"

//This Kernel resets all network states
template<typename T>
__global__ void reset_all_network_states_kernel(T* A, T* B, T* C, T*D, int n) {  // n is size

        //Addition by reading from Global Memory

	
	int thread_index = (blockIdx.x)*(blockDim.x) + threadIdx.x;

	if(thread_index < n) {
           A[thread_index] = 0;;
           B[thread_index] = 0;;
           C[thread_index] = 0;;
           D[thread_index] = 0;;
	}
}
 void LSTM::reset_all_network_states_gpu()
{
    //max threads per block needs to taken care while using dim3. Else kernel wont launch.
    dim3 dimBlock(m_block_dim*m_block_dim, 1, 1);
	dim3 dimGrid(((m_batch_size*m_hidden_size + m_block_dim*m_block_dim -1)/m_block_dim*m_block_dim), 1, 1); // There may be some incomplete(not block_dimXblock_dim) blocks at the edges
	//kernel Launch
	reset_all_network_states_kernel<float><<<dimGrid, dimBlock>>>(c_t_gpu, h_t_gpu, c_t_minus_1_gpu, h_t_minus_1_gpu, m_batch_size*m_hidden_size);
	     hipDeviceSynchronize();
}
