#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "LSTM.cuh"
using namespace std;

//Tiling is done across shared memories present in all SMs. 
//Each shared memory will get m_block_dim X m_block_dim amount of data each of A, B and compute C tile corresponding to them.
//standard (m,n,k) = (p,r,q) : k/q dimension is accumulating dimension for each output tile.
template<typename T>
__global__ void mat_mul_kernel(const T* A, const T* B, T*C, unsigned int p, unsigned int q, unsigned int r, int accum_mode){

if(threadIdx.x == 0)
   printf("kernel call mat_mul x : %d, y : %d, blockdim.x = %d, blockdim.y = %d\n", threadIdx.x, threadIdx.y, blockDim.x, blockDim.y);

//Dynamic shared memory declared
extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
T *shared_mem = reinterpret_cast<T *>(my_smem);

//assigning shared memory to tiled A,B,C pointers
T* tileA = (T*)shared_mem;
T* tileB = (T*)&shared_mem[blockDim.x*blockDim.y];
//T* tileC = (T*)&s[2*blockDim.x*blockDim.y];
T dot_sum = 0;
for(unsigned int i=0; i<((q+blockDim.x-1)/blockDim.x); i++)
{
	unsigned long int A_index = blockIdx.y*q*blockDim.y+i*blockDim.x+threadIdx.x+threadIdx.y*q;
	unsigned int A_index_x = i*blockDim.x+threadIdx.x;
	unsigned int A_index_y = blockIdx.y*blockDim.y + threadIdx.y;
//	if((A_index < p*q) && (A_index_x < q) && (A_index_y < p))
	    tileA[threadIdx.x+blockDim.x*threadIdx.y] = A[A_index];
//    else 
	    tileA[threadIdx.x+blockDim.x*threadIdx.y] = 0;

	//printf("tile A thread x : %d, thread y : %d, value : %f\n", threadIdx.x, threadIdx.y,  tileA[threadIdx.x+blockDim.x*threadIdx.y]);
	
	unsigned long int B_index = blockIdx.x*blockDim.x+i*r*blockDim.y+threadIdx.x+threadIdx.y*r;
    unsigned int B_index_x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int B_index_y = i*blockDim.y + threadIdx.y;
//	if((B_index < q*r) && (B_index_x < r) && (B_index_y < q))
	    tileB[threadIdx.x+blockDim.x*threadIdx.y] = B[B_index];
  //  else
	    tileB[threadIdx.x+blockDim.x*threadIdx.y] = 0;
    //printf("tile B thread x : %d, thread y : %d, value : %f\n", threadIdx.x, threadIdx.y,  tileB[threadIdx.x+blockDim.x*threadIdx.y]);

	//sync to make sure data is reached to shared memory
	__syncthreads();

   

	for(int k =0; k<blockDim.x; k++)
	{
         dot_sum += tileA[k+threadIdx.y*blockDim.x]*tileB[threadIdx.x + k*blockDim.x];
	 printf("threadId : %d, A = %f, B= %f\n", threadIdx.x, tileA[k+threadIdx.y*blockDim.x], tileB[threadIdx.x + k*blockDim.x]); 
	}
	//printf("tile C thread x : %d, thread y : %d, value : %f\n", threadIdx.x, threadIdx.y, dot_sum);
	__syncthreads();
}
    
	unsigned long int C_index = blockIdx.y*blockDim.y*r+threadIdx.y*r+blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int C_index_x = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int C_index_y = blockIdx.y*blockDim.y+threadIdx.y;
    if((C_index < p*r) && (C_index_x < r) && (C_index_y < p)) {
	        if(accum_mode == 1) {
		   C[C_index] += dot_sum;
		} else {
		   C[C_index] = dot_sum;
		}
		printf("threadId : %d, C[%d] = %f\n", threadIdx.x, C_index, C[C_index]);
	} 

}


void LSTM::mat_mul_gpu(const float *A, const float *B, float *C, int p, int q, int r){

    printf("gpu about to call kernel1 , block_dim : %d, p : %d, q : %d, r : %d\n", m_block_dim, p, q, r);
    //max threads per block needs to taken care while using dim3. Else kernel wont launch.
    dim3 dimBlock(m_block_dim, m_block_dim, 1);
	dim3 dimGrid(((p+dimBlock.x-1)/dimBlock.x), ((r+dimBlock.y-1)/dimBlock.y), 1); // There may be some incomplete(not block_dimXblock_dim) blocks at the edges
	//kernel Launch
	mat_mul_kernel<float><<<dimGrid, dimBlock, 2*(m_block_dim*m_block_dim)*sizeof(float)>>>(A, B, C, p, q, r, 0);
	//mat_mul_kernel<float><<<dimGrid, dimBlock>>>(A, B, C, p, q, r, 0);
    hipDeviceSynchronize();
    printf("gpu about to call kernel2 , block_dim : %d gridDim.x = %d, gridDim.y = %d\n", dimBlock.x, dimGrid.x, dimGrid.y);
}

