#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "LSTM.cuh"
__global__ void mat_add_kernel(const float* A, const float* B, float* C, int p, int q) {

        //Addition by reading from Global Memory
printf("kernel call mat_add_kernel : %d, blockdim.x = %d, blockdim.y = %d\n", threadIdx.x, blockDim.x, blockDim.y);

	int A_index = (blockDim.y)*(blockIdx.y)*q + (threadIdx.y)*q + (blockDim.x)*(blockIdx.x) + threadIdx.x;
	int B_index = (blockDim.y)*(blockIdx.y)*q + (threadIdx.y)*q + (blockDim.x)*(blockIdx.x) + threadIdx.x;
	int C_index = (blockDim.y)*(blockIdx.y)*q + (threadIdx.y)*q + (blockDim.x)*(blockIdx.x) + threadIdx.x;

        C[C_index] = A[A_index] + B[B_index];

}

__host__ void LSTM::mat_add_gpu(const float *A, const float *B, float *C, int p, int q){

    //max threads per block needs to taken care while using dim3. Else kernel wont launch.
    dim3 dimBlock(m_block_dim, m_block_dim, 1);
	dim3 dimGrid(((p+m_block_dim-1)/m_block_dim), ((q+m_block_dim-1)/m_block_dim), 1); // There may be some incomplete(not block_dimXblock_dim) blocks at the edges
	//kernel Launch
	mat_add_kernel<<<dimGrid, dimBlock>>>(A, B, C, p, q);
	     hipDeviceSynchronize();

}
