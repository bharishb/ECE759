#include "hip/hip_runtime.h"
#include <math.h>
#include <hip/hip_runtime.h>
#include "LSTM.cuh"

template<typename T>
__global__ void mat_sgm_kernel(const T* A, T* C, int p, int q) {

        //Addition by reading from Global Memory
printf("kernel call sgm : %d, blockdim.x = %d, blockdim.y = %d\n", threadIdx.x, blockDim.x, blockDim.y);

	int A_index = (blockIdx.x)*(blockDim.x)+ threadIdx.x;
	int C_index = (blockIdx.x)*(blockDim.x)+ threadIdx.x;

        C[C_index] = 1/(1 + expf(-A[A_index]));

}

__host__ void LSTM::mat_sgm_gpu(const float *A, float *C, int p, int q){

    printf("Sgm kernel before start\n");
    //max threads per block needs to taken care while using dim3. Else kernel wont launch.
    dim3 dimBlock(m_block_dim*m_block_dim, 1);
	dim3 dimGrid(((p*q+m_block_dim*m_block_dim-1)/(m_block_dim*m_block_dim)), 1, 1); // There may be some incomplete(not block_dimXblock_dim) blocks at the edges
	//kernel Launch
	mat_sgm_kernel<float><<<dimGrid, dimBlock>>>(A, C, p, q);
	     hipDeviceSynchronize();
    printf("Sgm kernel after end \n");

}
