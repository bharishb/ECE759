#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "LSTM.cuh"

//This Kernel removes stride accesses in input by moving data to a buffer.
__global__ void buffer_batch_input_kernel(const float* A, float* C, int offset, int n1, int n2, int stride) {  // n1 is Batch size, n2 is input size : extracting batch size inputs of each input_size amount at seq_length strides

        //Addition by reading from Global Memory

	
	int thread_index = (blockIdx.x)*(blockDim.x) + threadIdx.x;
	int A_index = (thread_index/n2)*stride + thread_index % n2 + offset;
	int C_index = thread_index;

	if((A_index < (n1*stride + n2)) && (C_index <n1*n2))  // need to handle input size
           C[C_index] = A[A_index];
    printf("Buffer kernel code threadIdx : %d, A[%d] = %f , C[%d] = %f\n", threadIdx.x, A_index, A[A_index], C_index, C[C_index]);
}

void LSTM::move_gpu_input_data_to_buffer(const float *A, int offset, float *C)
{
    printf("Buffer kernel before start\n");
    //max threads per block needs to taken care while using dim3. Else kernel wont launch.
    dim3 dimBlock(m_block_dim*m_block_dim, 1, 1);
	dim3 dimGrid(((m_batch_size*m_input_size + m_block_dim*m_block_dim -1)/(m_block_dim*m_block_dim)), 1, 1); // There may be some incomplete(not block_dimXblock_dim) blocks at the edges
	//kernel Launch
	buffer_batch_input_kernel<<<dimGrid, dimBlock>>>(A, C, offset, m_batch_size, m_input_size, m_seq_length*m_input_size);
    hipDeviceSynchronize();
    printf("Buffer kernel after end\n");
}
