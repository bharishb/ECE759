#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "LSTM.cuh"
template<typename T>
__global__ void mat_hadamard_mac_kernel(const T* A, const T* B, T* C, int p, int q, int accum_mode) {

        //Addition by reading from Global Memory

printf("kernel call hadamard mac: %d, blockdim.x = %d, blockdim.y = %d\n", threadIdx.x, blockDim.x, blockDim.y);
	int A_index = (blockDim.y)*(blockIdx.y)*q + (threadIdx.y)*q + (blockDim.x)*(blockIdx.x) + threadIdx.x;
	int B_index = (blockDim.y)*(blockIdx.y)*q + (threadIdx.y)*q + (blockDim.x)*(blockIdx.x) + threadIdx.x;
	int C_index = (blockDim.y)*(blockIdx.y)*q + (threadIdx.y)*q + (blockDim.x)*(blockIdx.x) + threadIdx.x;

	if(accum_mode == 1)
            C[C_index] += A[A_index] * B[B_index];
	else
            C[C_index] = A[A_index] * B[B_index];

}

__host__ void LSTM::mat_hadamard_mac_gpu(const float *A, const float *B, float *C, int p, int q){

    //max threads per block needs to taken care while using dim3. Else kernel wont launch.
    dim3 dimBlock(m_block_dim, m_block_dim, 1);
	dim3 dimGrid(((p+dimBlock.x-1)/dimBlock.x), ((q+dimBlock.y-1)/dimBlock.y), 1); // There may be some incomplete(not block_dimXblock_dim) blocks at the edges
	//kernel Launch
	mat_hadamard_mac_kernel<float><<<dimGrid, dimBlock>>>(A, B, C, p, q, 1);
	     hipDeviceSynchronize();

}
