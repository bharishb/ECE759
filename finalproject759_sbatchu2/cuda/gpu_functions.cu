#include<hip/hip_runtime.h>
#include "LSTM.cuh"
void LSTM::move_params_to_gpu(){
    printf("Moving params to GPU\n");
    hipMalloc((void**)&W_ii_gpu, sizeof(float)*m_input_size*m_hidden_size);
    hipMemcpy(W_ii_gpu, W_ii, m_input_size*m_hidden_size*(sizeof(float)), hipMemcpyHostToDevice); 
    hipMalloc((void**)&W_if_gpu, sizeof(float)*m_input_size*m_hidden_size);
    hipMemcpy(W_if_gpu, W_if, m_input_size*m_hidden_size*(sizeof(float)), hipMemcpyHostToDevice); 
    hipMalloc((void**)&W_ig_gpu, sizeof(float)*m_input_size*m_hidden_size);
    hipMemcpy(W_ig_gpu, W_ig, m_input_size*m_hidden_size*(sizeof(float)), hipMemcpyHostToDevice); 
    hipMalloc((void**)&W_io_gpu, sizeof(float)*m_input_size*m_hidden_size);
    hipMemcpy(W_io_gpu, W_io, m_input_size*m_hidden_size*(sizeof(float)), hipMemcpyHostToDevice); 

    
    hipMalloc((void**)&b_ii_gpu, sizeof(float)*m_hidden_size);
    hipMemcpy(b_ii_gpu, b_ii, m_hidden_size*(sizeof(float)), hipMemcpyHostToDevice); 
    hipMalloc((void**)&b_if_gpu, sizeof(float)*m_hidden_size);
    hipMemcpy(b_if_gpu, b_if, m_hidden_size*(sizeof(float)), hipMemcpyHostToDevice); 
    hipMalloc((void**)&b_ig_gpu, sizeof(float)*m_hidden_size);
    hipMemcpy(b_ig_gpu, b_ig, m_hidden_size*(sizeof(float)), hipMemcpyHostToDevice); 
    hipMalloc((void**)&b_io_gpu, sizeof(float)*m_hidden_size);
    hipMemcpy(b_io_gpu, b_io, m_hidden_size*(sizeof(float)), hipMemcpyHostToDevice); 

    hipMalloc((void**)&b_iiB_gpu, sizeof(float)*m_batch_size*m_hidden_size);
    hipMemcpy(b_iiB_gpu, b_iiB, m_batch_size*m_hidden_size*(sizeof(float)), hipMemcpyHostToDevice); 
    hipMalloc((void**)&b_ifB_gpu, sizeof(float)*m_batch_size*m_hidden_size);
    hipMemcpy(b_ifB_gpu, b_ifB, m_batch_size*m_hidden_size*(sizeof(float)), hipMemcpyHostToDevice); 
    hipMalloc((void**)&b_igB_gpu, sizeof(float)*m_batch_size*m_hidden_size);
    hipMemcpy(b_igB_gpu, b_igB, m_batch_size*m_hidden_size*(sizeof(float)), hipMemcpyHostToDevice); 
    hipMalloc((void**)&b_ioB_gpu, sizeof(float)*m_batch_size*m_hidden_size);
    hipMemcpy(b_ioB_gpu, b_ioB, m_batch_size*m_hidden_size*(sizeof(float)), hipMemcpyHostToDevice); 

    hipMalloc((void**)&W_hi_gpu, sizeof(float)*m_hidden_size*m_hidden_size);
    hipMemcpy(W_hi_gpu, W_hi, m_hidden_size*m_hidden_size*(sizeof(float)), hipMemcpyHostToDevice); 
    hipMalloc((void**)&W_hf_gpu, sizeof(float)*m_hidden_size*m_hidden_size);
    hipMemcpy(W_hf_gpu, W_hf, m_hidden_size*m_hidden_size*(sizeof(float)), hipMemcpyHostToDevice); 
    hipMalloc((void**)&W_hg_gpu, sizeof(float)*m_hidden_size*m_hidden_size);
    hipMemcpy(W_hg_gpu, W_hg, m_hidden_size*m_hidden_size*(sizeof(float)), hipMemcpyHostToDevice); 
    hipMalloc((void**)&W_ho_gpu, sizeof(float)*m_hidden_size*m_hidden_size);
    hipMemcpy(W_ho_gpu, W_ho, m_hidden_size*m_hidden_size*(sizeof(float)), hipMemcpyHostToDevice); 

    hipMalloc((void**)&b_hi_gpu, sizeof(float)*m_hidden_size);
    hipMemcpy(b_hi_gpu, b_hi, m_hidden_size*(sizeof(float)), hipMemcpyHostToDevice); 
    hipMalloc((void**)&b_hf_gpu, sizeof(float)*m_hidden_size);
    hipMemcpy(b_hf_gpu, b_hf, m_hidden_size*(sizeof(float)), hipMemcpyHostToDevice); 
    hipMalloc((void**)&b_hg_gpu, sizeof(float)*m_hidden_size);
    hipMemcpy(b_hg_gpu, b_hg, m_hidden_size*(sizeof(float)), hipMemcpyHostToDevice); 
    hipMalloc((void**)&b_ho_gpu, sizeof(float)*m_hidden_size);
    hipMemcpy(b_ho_gpu, b_ho, m_hidden_size*(sizeof(float)), hipMemcpyHostToDevice); 

    hipMalloc((void**)&b_hiB_gpu, sizeof(float)*m_batch_size*m_hidden_size);
    hipMemcpy(b_hiB_gpu, b_hiB, m_batch_size*m_hidden_size*(sizeof(float)), hipMemcpyHostToDevice); 
    hipMalloc((void**)&b_hfB_gpu, sizeof(float)*m_batch_size*m_hidden_size);
    hipMemcpy(b_hfB_gpu, b_hfB, m_batch_size*m_hidden_size*(sizeof(float)), hipMemcpyHostToDevice); 
    hipMalloc((void**)&b_hgB_gpu, sizeof(float)*m_batch_size*m_hidden_size);
    hipMemcpy(b_hgB_gpu, b_hgB, m_batch_size*m_hidden_size*(sizeof(float)), hipMemcpyHostToDevice); 
    hipMalloc((void**)&b_hoB_gpu, sizeof(float)*m_batch_size*m_hidden_size);
    hipMemcpy(b_hoB_gpu, b_hoB, m_batch_size*m_hidden_size*(sizeof(float)), hipMemcpyHostToDevice); 

    hipMalloc((void**)&h_t_gpu, sizeof(float)*m_batch_size*m_hidden_size);
    hipMalloc((void**)&h_t_minus_1_gpu, sizeof(float)*m_batch_size*m_hidden_size);
    hipMalloc((void**)&c_t_gpu, sizeof(float)*m_batch_size*m_hidden_size);
    hipMalloc((void**)&c_t_minus_1_gpu, sizeof(float)*m_batch_size*m_hidden_size);

    hipMalloc((void**)&W_nn_gpu, sizeof(float)*m_hidden_size);
    hipMalloc((void**)&b_nn_gpu, sizeof(float));
    hipMalloc((void**)&b_nnB_gpu, sizeof(float)*m_batch_size);

    hipMalloc((void**)&m_gpu_buff_ptr, sizeof(float)*m_batch_size*m_input_size);
    //hipDeviceSynchronize();
}

void LSTM::move_inputs_to_gpu(const float* x, int num_inputs){
    printf("Moving Inputs to GPU\n");
    hipMalloc((void**)&m_gpu_input_ptr, sizeof(float)*num_inputs*m_seq_length*m_input_size);
    hipMemcpy(m_gpu_input_ptr, x, num_inputs*m_seq_length*m_input_size*(sizeof(float)), hipMemcpyHostToDevice); 
    hipMalloc((void**)&m_gpu_output_ptr, sizeof(float)*num_inputs);
    //hipDeviceSynchronize();
}

void LSTM::move_outputs_to_cpu(float* x, int num_inputs){
    hipMemcpy(x, m_gpu_output_ptr, num_inputs*(sizeof(float)), hipMemcpyDeviceToHost); 
    //hipDeviceSynchronize();
}
