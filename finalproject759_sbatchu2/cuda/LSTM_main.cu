#include <stdio.h>
#include <iostream>
#include <fstream>
#include "LSTM.cuh"
#include <hip/hip_runtime.h>

using namespace std;

void load_data(float* data, const char* filename){
   //printf("Loading Data from %s\n",filename); 
   ifstream in(filename);
   int i=0;
   while(!in.eof()) {
      in>>data[i];
      i++;
   }
}

void dump_data(float* data, int length, const char* filename){
   printf("Dumping Data To %s\n",filename);
   ofstream out(filename);
 if(out.is_open()){	  
   for(int i=0; i<length; i++) {
              out<<data[i];
	      out<<",\n";
	   }
 }
   out.close();
}

void print_data(float* data, int length){
	printf("Printing Data\n");
	for(int i=0; i<length; i++)
		printf("data[%d] = %f\n", i, data[i]);
    
}

int main(int argc, char** argv){


    int input_size = 1;
    int hidden_size = 2;
    int seq_length = 4;
    int num_inputs = 139;
    //int num_inputs = 1;
    int batch_size = atoi(argv[1]);
    char* device = argv[2]; // cpu, gpu
    int block_dim = atoi(argv[3]); // 32 : blockDim
    char* device_language = argv[4]; //cpp, cuda, openmp, mpi
    int N_iter = 1000;

    printf("Running GPU Implementation of LSTM\n");

    LSTM LSTM_inst(input_size, hidden_size, seq_length, batch_size, "weights.txt", device, block_dim, device_language);

    LSTM_inst.load_weights();
    //LSTM_inst.print_weights();

    float* inputs = new float[seq_length*num_inputs*N_iter];
    float* outputs = new float[num_inputs*N_iter];
    for(int i=0; i<N_iter; i++)
        load_data(inputs + i*seq_length*num_inputs,"LSTM_inputs.txt");
    //print_data(inputs, seq_length*num_inputs);
//LSTM call with timing
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    LSTM_inst.forward_pass(inputs, outputs, num_inputs*N_iter);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    //print_data(outputs, num_inputs);
    //dump_data(outputs, num_inputs*N_iter, "outputs.txt");

    float ms;
    hipEventElapsedTime(&ms, start, stop);
    printf("%f\n", ms);  
    return 0;
}
