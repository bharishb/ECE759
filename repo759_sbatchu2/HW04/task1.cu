#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "matmul.cuh"
#include <stdio.h>
#include <iostream>
#include <random>
#include <cassert>
using namespace std;

void print_matrix(const float *X, int n) {
  cout << "Matrix check" << endl;
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      cout << (int)(X[j + i * n]*1000) << " ";
    }
    cout << endl;
  } 
  cout << endl;
}   

void matmul_golden(const float*A, const float* B, float*C, size_t n){
    for(size_t i=0; i<n; i++){
	    for(size_t k=0; k<n; k++){
		    for(size_t j=0; j<n; j++){
                        if (k == 0) {
                          C[j + i * n] = 0; // initialization
                        }
                        C[j + i * n] = C[j + i * n] + A[i * n + k] * B[k * n + j]; // dot
		    
		    }
	    }
    }

}

void compare_matrix(const float* A, const float* B, size_t n)
{
    for(size_t i=0; i<n; i++) {
	    for(size_t j=0; j<n; j++) {
		    assert(abs(A[i+j*n]-B[i+j*n])<0.002);
	            /*if(((int)(A[i+j*n]*100)!=(int)(B[i+j*n]*100))) {
		           printf("MISMATCH golden value = %f, observed value = %f\n",A[i+j*n], B[i+j*n]);
		    }*/
	    }
    }
}

int main(int argc, char** argv)
{
    size_t n = atoi(argv[1]);
    unsigned int threads_per_block = atoi(argv[2]);

    float* hA;
    float* hB;
    float* hC;
    //float* hC_golden;

    hA = new float[n*n]; 
    hB = new float[n*n]; 
    hC = new float[n*n]; 
    //hC_golden = new float[n*n]; 

    //random number generator
    std::random_device rd;
    std::mt19937 generator(rd());
    std::uniform_real_distribution<float> dista(-1,1);
    std::uniform_real_distribution<float> distb(-1,1);

    for(size_t i=0; i<n*n; i++)
    {
        hA[i] = dista(generator);
        hB[i] = distb(generator);
    }

    //device memory
    float* dA;
    float* dB;
    float* dC;   //device memory

    //Allocate device memory
    hipMalloc((void**)&dA, sizeof(float)*n*n);
    hipMalloc((void**)&dB, sizeof(float)*n*n);
    hipMalloc((void**)&dC, sizeof(float)*n*n);

    //Move to device memory
    hipMemcpy(dA, hA, sizeof(float)*n*n, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, sizeof(float)*n*n, hipMemcpyHostToDevice);

    //Kernel call with timing
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    matmul(dA, dB, dC, n, threads_per_block);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    //Move from device memory to host memory
    hipMemcpy(hC, dC, sizeof(float)*n*n, hipMemcpyDeviceToHost);


    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    //printf("Time Taken in Milliseconds : %f",ms);
    //last element print
    printf("%f\n",hC[n*n-1]);
    printf("%f\n", ms);
    //matmul_golden(hA, hB, hC_golden, n);
    //compare_matrix(hC, hC_golden, n);
    /*print_matrix(hA,n);
    print_matrix(hB,n);
    print_matrix(hC,n);
    print_matrix(hC_golden,n);*/
    delete [] hA;
    hA = nullptr;
    delete [] hB;
    hB = nullptr;
    delete [] hC;
    hC = nullptr;

    return 0;
}
