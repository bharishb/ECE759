#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "stencil.cuh"
#include <iostream>
using namespace std;

__global__ void stencil_kernel(const float* image, const float* mask, float* output, unsigned int n, unsigned int R){


	//shared memory
	extern __shared__ float s[];
	float* shared_mask = s;   // 2*R+1 size

	//Let say thread block starts at index x1 and ends at x2. image values needed are from x1-R to x2+R. That is x2-x1+2R+1 size. That is blockDim.x+2R size
        float* image_thread_block = (float*)&shared_mask[2*R+1]; //threads_per_block + 2*R size
	float* output_thread_block = (float*)&image_thread_block[blockDim.x + 2*R]; //threads_per_block

	if(threadIdx.x < (2*R +1)) {
	    shared_mask[threadIdx.x] = mask[threadIdx.x];
	}

	
        int index1 = blockDim.x*blockIdx.x - R;  // x1 = blockDim1.x*blockIdx.x is the starting index
	for(int k =0; (k*blockDim.x+threadIdx.x)<(blockDim.x + 2*R); k++) {  // Some threads of the block may need to pull more than 1 data per thread. This is loop based of thread index pulling more than 1 data per thread.
		    int i = k*blockDim.x+threadIdx.x;
		    if(((index1 + i)<0) || ((index1 + i)>(n-1))) {
		    	image_thread_block[i] = 1;
		    } else {
		    	image_thread_block[i] = image[index1+i];  //x1 - R to x2 + R : x1, x2 are start and end indices of thread block
		    }
	}
       
        	

        __syncthreads();
	
	if((blockDim.x*blockIdx.x + threadIdx.x)<n) {
	//output
	float output_conv = 0.0; // trying to write into local thread register than memory
        for(int j=0; j<(2*R+1) ; j++)
	{
	    output_conv += image_thread_block[j + threadIdx.x]*shared_mask[j];
	}
	output_thread_block[threadIdx.x] = output_conv;
	}

	__syncthreads();

	//writing back to global memory
	if((blockDim.x*blockIdx.x + threadIdx.x)<n) {
        output[blockDim.x*blockIdx.x + threadIdx.x] = output_thread_block[threadIdx.x];
	}
}


__host__ void stencil(const float* image, const float* mask, float* output, unsigned int n, unsigned int R, unsigned int threads_per_block){


      //kernel call
      stencil_kernel<<<(n+threads_per_block-1)/threads_per_block,threads_per_block,(2*R+1)*sizeof(float)+(2*R+threads_per_block)*sizeof(float)+(threads_per_block)*sizeof(float)>>>(image,mask, output, n, R);

}
