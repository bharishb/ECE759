#include "hip/hip_runtime.h"
#include "matmul.cuh"
#include <hip/hip_runtime.h>


__global__ void matmul_kernel(const float* A, const float* B, float* C, size_t n)
{
	size_t index = blockIdx.x*blockDim.x + threadIdx.x;
        
	//compute matrix output element

	if(index < n*n) {
		float sum = 0.0;
		for(size_t k=0; k <n; k++)
		{
		    sum = sum + A[index - index%n + k]*B[k*n + (index%n)];  // doing reduction on registers rather than memory
		}
		C[index] = sum; //writing final value in memory
	}
}

void matmul(const float* A, const float* B, float* C, size_t n, unsigned int threads_per_block)
{

	//call kernel
	matmul_kernel<<<(n*n+threads_per_block-1)/threads_per_block, threads_per_block>>>(A, B, C, n);  //kernel launch

}
