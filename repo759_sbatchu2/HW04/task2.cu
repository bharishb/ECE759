#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "stencil.cuh"
#include <stdio.h>
#include <iostream>
#include <random>
#include <cassert>
using namespace std;

void print_matrix(const float *X, int n) {
  cout << "Matrix check" << endl;
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      cout << X[j + i * n] << " ";
    }
    cout << endl;
  }
  cout << endl;
}

void print_vector(const float *X, int n){
   cout << "Vector check" << endl;
   for(int i=0; i<n; i++)
   {
       cout << X[i] << " ";
   } 
   cout << endl;
}

void stencil_golden(const float* a, const float*b, float*c, unsigned int n, unsigned int R)
{
    for(unsigned int i=0; i<n; i++)
    { 
	    c[i] = 0.0;
	    for(unsigned int j=0; j<(2*R+1); j++) {
                int index = i + j - R;
	        c[i] += (((index<0) || (index>(long int)(n-1))) ? 1 : a[index]) *b[j];
	    }
    }

}	

void compare_vector(const float* X, const float* Y, unsigned int n)
{ 
	for(unsigned int i=0; i<n; i++)
	{
            assert(abs(X[i]-Y[i])<0.002);
	    /*if(abs(X[i]-Y[i])>0.01)
	    {
	        printf("Mismatch seen. X[%d]=%f, Y[%d]=%f\n",i, X[i], i, Y[i]);
	    }*/
	}
}

int main(int argc, char** argv)
{
    size_t n = atoi(argv[1]);
    size_t R = atoi(argv[2]);
    unsigned int threads_per_block = atoi(argv[3]);

    float* h_image;
    float* h_mask;
    float* h_output;
    float* h_output_golden;

    h_image = new float[n]; 
    h_mask = new float[2*R+1]; 
    h_output = new float[n]; 
    h_output_golden = new float[n]; 

    //random number generator
    std::random_device rd;
    std::mt19937 generator(rd());
    std::uniform_real_distribution<float> dist_image(-1,1);
    std::uniform_real_distribution<float> dist_mask(-1,1);

    for(size_t i=0; i<n; i++)
    {
        h_image[i] = dist_image(generator);
    }

    for(size_t i=0; i<(2*R+1); i++)
    {
        h_mask[i] = dist_mask(generator);
    }


    float* d_image;
    float* d_mask;
    float* d_output;

    //global memory allocation
    hipMalloc((void**)&d_image, sizeof(float)*n);
    hipMalloc((void**)&d_mask, sizeof(float)*(2*R+1));
    hipMalloc((void**)&d_output, sizeof(float)*n);
    
    //moving data to device
    hipMemcpy(d_image, h_image, sizeof(float)*n, hipMemcpyHostToDevice);
    hipMemcpy(d_mask, h_mask, sizeof(float)*(2*R+1), hipMemcpyHostToDevice);

    //Kernel call with timing
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    stencil(d_image, d_mask, d_output, n, R, threads_per_block);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipMemcpy(h_output, d_output, sizeof(float)*n, hipMemcpyDeviceToHost);

    hipFree(d_image);
    hipFree(d_mask);
    hipFree(d_output);

    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    //printf("Time Taken in Milliseconds : %f",ms);
    //last element print
    printf("%f\n",h_output[n-1]);
    printf("%f\n", ms);

    //check 
    stencil_golden(h_image, h_mask, h_output_golden, n, R);
    compare_vector(h_output_golden, h_output, n);

    delete [] h_image;
    h_image = nullptr;
    delete [] h_mask;
    h_mask = nullptr;
    delete [] h_output;
    h_output = nullptr;

    return 0;

}
