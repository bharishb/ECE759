#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "reduce.cuh"
#include <stdio.h>

__global__ void reduce_kernel(float *g_idata, float *g_odata, unsigned int n){
    extern __shared__ float sdata[];
    unsigned int index = threadIdx.x + 2*blockDim.x*blockIdx.x;
    //printf("thread index : %d , blockIdx : %d, blockDim : %d, n : %d, index : %d, value : %f, input1 : %f input2 : %f\n", threadIdx.x , blockIdx.x, blockDim.x, n, index, sdata[threadIdx.x], g_idata[index],g_idata[index + blockDim.x]);
    if(index < n) {
        if((index + blockDim.x)<n){
            sdata[threadIdx.x] = g_idata[index] + g_idata[index + blockDim.x];
        } else{
            sdata[threadIdx.x] = g_idata[index];
        }
    } else {
        sdata[threadIdx.x] = 0.0;
    }
    __syncthreads(); //making sure data is available in shared memory

    //shared mem print
    //printf("thread index : %d , blockIdx : %d , value : %f, input1 : %f input2 : %f\n", threadIdx.x , blockIdx.x, sdata[threadIdx.x], g_idata[index],g_idata[index + blockDim.x]);

    //reduce in shared memory - Tried handling non 2 power block sizes
    for(unsigned int s=blockDim.x; s>1; s=(s+1)/2){
        if(threadIdx.x < (s)/2){
            sdata[threadIdx.x] += sdata[threadIdx.x+(s+1)/2];
        }
        __syncthreads();
    }
    g_odata[blockIdx.x] = sdata[0];
}


__host__ void reduce(float **input, float **output, unsigned int N, unsigned int threads_per_block)
{
   //launch multiple times to ensure 1 thread block left for execution
   unsigned int i;
   for(i = N; i>2*threads_per_block; i=(i+2*threads_per_block-1)/(2*threads_per_block)){
    //printf("host function calling kernel i : %d\n", i);
       reduce_kernel<<<(i+2*threads_per_block-1)/(2*threads_per_block), threads_per_block, threads_per_block*sizeof(float)>>>(*input, *output, i);        
   }

    reduce_kernel<<<1, threads_per_block, threads_per_block*sizeof(float)>>>(*input, *output, i);        

}