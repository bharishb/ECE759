#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "reduce.cuh"
#include <stdio.h>
#include <iostream>
#include <random>
#include <cassert>
using namespace std;

void print_vector(const float *X, size_t n) {
  cout << "Vector check" << endl;
    for (size_t j = 0; j < n; j++) {
      cout << X[j] << " ";
    }
    cout << endl;
}   

void reduce_golden(const float* X, float* sum, unsigned int n){
    *sum =0.0;
    for(unsigned int i=0; i<n; i++){
        *sum+=X[i];
    }
}

int main(int argc, char** argv){

unsigned int n = atoi(argv[1]);
unsigned int threads_per_block = atoi(argv[2]);
    
float* h_input;
float* d_input;
float sum_golden;

h_input = new float[n];

//random number generator
std::random_device rd;
std::mt19937 generator(rd());

std::uniform_real_distribution<float> dist(-1,1);

for(size_t i=0; i<n; i++)
{
    h_input[i] = dist(generator);
}

//golden
//print_vector(h_input, n);
reduce_golden(h_input,&sum_golden, n);
//cout<<"sum :"<<sum_golden<<endl;

//Allocate device memory
hipMalloc((void**)&d_input, sizeof(float)*n);

//Move to device memory
hipMemcpy(d_input, h_input, sizeof(float)*n, hipMemcpyHostToDevice);

//Kernel call with timing
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    reduce(&d_input, &d_input, n, threads_per_block);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    //Move from device memory to host memory
    hipMemcpy(h_input, d_input, sizeof(float)*n, hipMemcpyDeviceToHost);

    hipFree(d_input);

    //GPU sum
    //cout <<"GPU Sum : "<<h_input[0]<<endl;
    cout <<h_input[0]<<endl;
    assert(abs(sum_golden-h_input[0])<0.5);

    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    printf("%f\n", ms);  
    //printf("Time Taken in Milliseconds : %f",ms);

    delete [] h_input;
    h_input = nullptr;

return 0;

}

