#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "matmul.cuh"
#include <stdio.h>
#include <iostream>
#include <random>
#include <cassert>
using namespace std;

template<typename T>
void print_matrix(const T *X, size_t n) {
  cout << "Matrix check" << endl;
  for (size_t i = 0; i < n; i++) {
    for (size_t j = 0; j < n; j++) {
      cout << X[j + i * n] << " ";
    }
    cout << endl;
  } 
  cout << endl;
}   

template<typename T>
void matmul_golden(const T*A, const T* B, T*C, size_t n){
    for(size_t i=0; i<n; i++){
	    for(size_t k=0; k<n; k++){
		    for(size_t j=0; j<n; j++){
                        if (k == 0) {
                          C[j + i * n] = 0; // initialization
                        }
                        C[j + i * n] = C[j + i * n] + A[i * n + k] * B[k * n + j]; // dot
		    
		    }
	    }
    }

}

template<typename T>
void compare_matrix(const T* A, const T* B, size_t n)
{
    for(size_t i=0; i<n; i++) {
	    for(size_t j=0; j<n; j++) {
		    assert(abs(A[i+j*n]-B[i+j*n])<5);
		    /*if(abs(A[i+j*n]-B[i+j*n])>5) {
		           printf("MISMATCH golden value = %f, observed value = %f\n",A[i+j*n], B[i+j*n]);
		     return;}*/
	    }
    }
}


template<typename T>
void launch_matmul_kernel(size_t n, unsigned int block_dim, double* hA_double, double* hB_double){

    T* hA;
    T* hB;
    T* hC;
    T* hC_golden;

    hA = new T[n*n]; 
    hB = new T[n*n]; 
    hC = new T[n*n]; 
    hC_golden = new T[n*n];


    for(unsigned long long int i=0; i<n*n ; i++){
        hA[i] = (T) hA_double[i];
        hB[i] = (T) hB_double[i];
    }

    //device memory
    //matmul_1
    T* dA;
    T* dB;
    T* dC;   //device memory

    //Allocate device memory
    hipMalloc((void**)&dA, sizeof(T)*n*n);
    hipMalloc((void**)&dB, sizeof(T)*n*n);
    hipMalloc((void**)&dC, sizeof(T)*n*n);

    //Move to device memory
    hipMemcpy(dA, hA, sizeof(T)*n*n, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, sizeof(T)*n*n, hipMemcpyHostToDevice);

    //Kernel call with timing
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    if (std::is_same<T, int>::value){
        matmul_1((int*)dA, (int*)dB, (int*)dC, n, block_dim);
    } else if (std::is_same<T, float>::value){
        matmul_2((float*)dA, (float*)dB, (float*)dC, n, block_dim);
    } else {
        matmul_3((double*)dA, (double*)dB, (double*)dC, n, block_dim);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    //Move from device memory to host memory
    hipMemcpy(hC, dC, sizeof(T)*n*n, hipMemcpyDeviceToHost);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    //printf("Time Taken in Milliseconds : %f",ms);

    if (std::is_same<T, int>::value){
      //first element
      printf("%d\n",(int)hC[0]);
      //last element print
      printf("%d\n",(int)hC[n*n-1]);
    } else if (std::is_same<T, float>::value){
      //first element
      printf("%f\n",(float)hC[0]);
      //last element print
      printf("%f\n",(float)hC[n*n-1]);
    } else {
      //first element
      printf("%f\n",(double)hC[0]);
      //last element print
      printf("%f\n",(double)hC[n*n-1]);
    }
    printf("%f\n", ms);
    //matmul_golden<T>(hA, hB, hC_golden, n);
    //compare_matrix<T>(hC_golden, hC, n);
    /*print_matrix(hA,n);
    print_matrix(hB,n);
    print_matrix(hC,n);
    print_matrix(hC_golden,n);*/
    delete [] hA;
    hA = nullptr;
    delete [] hB;
    hB = nullptr;
    delete [] hC;
    hC = nullptr;
    delete [] hC_golden;
    hC_golden = nullptr;
}

int main(int argc, char** argv)
{
    size_t n = atoi(argv[1]);
    unsigned int block_dim = atoi(argv[2]);
   
    //trying to give same matrix data with various datatypes
    double* hA_double;
    double* hB_double;

    hA_double = new double[n*n];
    hB_double = new double[n*n];

    //random number generator
    std::random_device rd;
    std::mt19937 generator(rd());

    std::uniform_real_distribution<double> dista(-1000,1000);
    std::uniform_real_distribution<double> distb(-1000,1000);

    for(size_t i=0; i<n*n; i++)
    {
      hA_double[i] = dista(generator);
      hB_double[i] = distb(generator);
    }

    //matmul_1
    launch_matmul_kernel<int>(n, block_dim, hA_double, hB_double);

    //matmul_2
    launch_matmul_kernel<float>(n, block_dim, hA_double, hB_double);

    //matmul_3
    launch_matmul_kernel<double>(n, block_dim, hA_double, hB_double);
}
