#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "matmul.cuh"
#include <stdio.h>
#include <iostream>
using namespace std;

//Tiling is done across shared memories present in all SMs. 
//Each shared memory will get block_dim X block_dim amount of data each of A, B and compute C tile corresponding to them.
template<typename T>
__global__ void matmul_kernel(const T* A, const T* B, T*C, unsigned int n){

//Dynamic shared memory declared
extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
T *shared_mem = reinterpret_cast<T *>(my_smem);

//assigning shared memory to tiled A,B,C pointers
T* tileA = (T*)shared_mem;
T* tileB = (T*)&shared_mem[blockDim.x*blockDim.y];
//T* tileC = (T*)&s[2*blockDim.x*blockDim.y];

T dot_sum = 0;
for(unsigned int i=0; i<((n+blockDim.x-1)/blockDim.x); i++)
{
	unsigned long int A_index = blockIdx.y*n*blockDim.y+i*blockDim.x+threadIdx.x+threadIdx.y*n;
	unsigned int A_index_x = i*blockDim.x+threadIdx.x;
	unsigned int A_index_y = blockIdx.y*blockDim.y + threadIdx.y;
	if((A_index < n*n) && (A_index_x < n) && (A_index_y < n))
	    tileA[threadIdx.x+blockDim.x*threadIdx.y] = A[A_index];
    else 
	    tileA[threadIdx.x+blockDim.x*threadIdx.y] = 0;

	//printf("tile A thread x : %d, thread y : %d, value : %f\n", threadIdx.x, threadIdx.y,  tileA[threadIdx.x+blockDim.x*threadIdx.y]);
	
	unsigned long int B_index = blockIdx.x*blockDim.x+i*n*blockDim.y+threadIdx.x+threadIdx.y*n;
    unsigned int B_index_x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int B_index_y = i*blockDim.y + threadIdx.y;
	if((B_index < n*n) && (B_index_x < n) && (B_index_y < n))
	    tileB[threadIdx.x+blockDim.x*threadIdx.y] = B[B_index];
    else
	    tileB[threadIdx.x+blockDim.x*threadIdx.y] = 0;
    //printf("tile B thread x : %d, thread y : %d, value : %f\n", threadIdx.x, threadIdx.y,  tileB[threadIdx.x+blockDim.x*threadIdx.y]);

	//sync to make sure data is reached to shared memory
	__syncthreads();

   

	for(int k =0; k<blockDim.x; k++)
	{
         dot_sum += tileA[k+threadIdx.y*blockDim.x]*tileB[threadIdx.x + k*blockDim.x];
	}
	//printf("tile C thread x : %d, thread y : %d, value : %f\n", threadIdx.x, threadIdx.y, dot_sum);
	__syncthreads();
}
    
	unsigned long int C_index = blockIdx.y*blockDim.y*n+threadIdx.y*n+blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int C_index_x = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int C_index_y = blockIdx.y*blockDim.y+threadIdx.y;
    if((C_index < n*n) && (C_index_x < n) && (C_index_y < n)) {
		C[C_index] = dot_sum;
	} 

}





__host__ void matmul_1(const int *A, const int *B, int *C, unsigned int n, unsigned int block_dim){

    dim3 dimBlock(block_dim, block_dim);
	dim3 dimGrid((n+block_dim-1)/block_dim, (n+block_dim-1)/block_dim); // There may be some incomplete(not block_dimXblock_dim) blocks at the edges

	//kernel Launch
	matmul_kernel<int><<<dimGrid, dimBlock, 2*(block_dim*block_dim)*sizeof(int)>>>(A, B, C, n);

}

__host__ void matmul_2(const float *A, const float *B, float *C, unsigned int n, unsigned int block_dim){

    //max threads per block needs to taken care while using dim3. Else kernel wont launch.
    dim3 dimBlock(block_dim, block_dim, 1);
	dim3 dimGrid(((n+dimBlock.x-1)/dimBlock.x), ((n+dimBlock.y-1)/dimBlock.y), 1); // There may be some incomplete(not block_dimXblock_dim) blocks at the edges
	//kernel Launch
	matmul_kernel<<<dimGrid, dimBlock, 2*(block_dim*block_dim)*sizeof(float)>>>(A, B, C, n);
}

__host__ void matmul_3(const double *A, const double *B, double *C, unsigned int n, unsigned int block_dim){

    dim3 dimBlock(block_dim, block_dim);
	dim3 dimGrid((n+block_dim-1)/block_dim, (n+block_dim-1)/block_dim); // There may be some incomplete(not block_dimXblock_dim) blocks at the edges

	//kernel Launch
	matmul_kernel<double><<<dimGrid, dimBlock, 2*(block_dim*block_dim)*sizeof(double)>>>(A, B, C, n);

}
