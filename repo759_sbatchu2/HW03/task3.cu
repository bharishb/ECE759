#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <random>
#include "vscale.cuh"
#define NUM_THREADS_PER_BLOCK 16
using namespace std;

int main(int argc, char**argv){
    char* n_string = argv[1]; // First argument, 0 is executable
    unsigned int  n = atoi(n_string);

    //random number generator
    std::random_device rd;
    std::mt19937 generator(rd());
    std::uniform_real_distribution<float> dista(-10.0,10.0);
    std::uniform_real_distribution<float> distb(0,1.0);

    float* host_a;
    float* host_b;
    float* device_a;
    float* device_b;
    host_a = new float[n];
    host_b = new float[n];

    for(unsigned int i=0; i<n; i++)
    {
        host_a[i] = dista(generator);
        host_b[i] = distb(generator);
    }

    /*//print input arrays
    printf("Array a\n");
    for(int i=0; i<n; i++) printf("%f ",host_a[i]);
    printf("\nArray b\n");
    for(int i=0; i<n; i++) printf("%f ",host_b[i]);;
    printf("\nArray Expected output\n");
    for(int i=0; i<n; i++) printf("%f ",host_b[i]*host_a[i]);
    printf("\n");*/

    //Allocate device Memory
    hipMalloc((void**)&device_a, sizeof(float)*n);
    hipMalloc((void**)&device_b, sizeof(float)*n);
    hipMemcpy(device_a, host_a, sizeof(float)*n, hipMemcpyHostToDevice);
    hipMemcpy(device_b, host_b, sizeof(float)*n, hipMemcpyHostToDevice);

    //Kernel call with timing
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    vscale<<<(n+NUM_THREADS_PER_BLOCK-1)/NUM_THREADS_PER_BLOCK,NUM_THREADS_PER_BLOCK>>>(device_a, device_b, n); // kernel called
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipMemcpy(host_b, device_b, sizeof(float)*n, hipMemcpyDeviceToHost);
    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    //printf("Time Taken in Milliseconds : %f",ms);
    printf("%f\n", ms);
    printf("%f\n",host_b[0]);
    printf("%f\n",host_b[n-1]);

    /*printf("Array b output \n");
    for(int i=0; i<n; i++) printf("%f ",host_b[i]);*/
    
    return 0;
}
