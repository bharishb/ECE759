#include "hip/hip_runtime.h"
#include "vscale.cuh"

__global__ void vscale(const float *a, float *b, unsigned int n)
{

    unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;

    //Avoid writing into unknown locations
    if(index < n){
        b[index] = b[index]*a[index];
    }
}
