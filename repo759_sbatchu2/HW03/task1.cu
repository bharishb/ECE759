#include <iostream>
#include<stdio.h>
#include <hip/hip_runtime.h>
using namespace std;

__global__ void factorial()
{
    int fac;
    for(int i=1; i<=(threadIdx.x+1); i++)
    {
        fac = (i==1) ? 1 : fac*i ;
    }
    printf("%d!=%d\n",(threadIdx.x+1),fac);
}
int main()
{
   
    factorial<<<1,8>>>(); // 1 block, 8 threads per block
    hipDeviceSynchronize();
    return 0;
}
