#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <random>
using namespace std;
#define NUM_THREADS_PER_BLOCK 8
#define NUM_ELEMS 16

__global__ void saxpy_int_example(int* dA, int a)
{
    int index = blockIdx.x*NUM_THREADS_PER_BLOCK + threadIdx.x;
    //Avoid writing into unknown locations
    if(index < NUM_ELEMS){
        dA[index] = a*threadIdx.x + blockIdx.x;
    }
}

int main()
{ 
    int hA[NUM_ELEMS];  //host array
    int* dA;            //Device array

    //random number generator
    std::random_device rd;
    std::mt19937 generator(rd());
    std::uniform_int_distribution<int> dist(-1000,1000);
    int a = dist(generator);

    hipMalloc((void**)&dA, sizeof(int)*NUM_ELEMS);
    saxpy_int_example<<<NUM_ELEMS/NUM_THREADS_PER_BLOCK, NUM_THREADS_PER_BLOCK>>>(dA,a); // calling the kernel with 2 thread blocks with 8 elements each
    hipMemcpy(hA, dA, NUM_ELEMS*(sizeof(int)), hipMemcpyDeviceToHost); // dont need explicit cudaDeviceSynchronize
    for(int i=0; i<NUM_ELEMS; i++){
        printf("%d ",hA[i]);
    }

}
