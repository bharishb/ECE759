#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <iostream>
#include <random>
#include <cassert>
using namespace std;

void reduce_golden(const float* X, float* sum, unsigned int n){
    *sum = 0.0;
    for(unsigned int i=0; i<n; i++){
        *sum+=X[i];
    }
}

int main(int argc, char** argv){
	
    unsigned int n = atoi(argv[1]);
    float sum;
    float sum_golden;
    //allocate host memory
    thrust::host_vector<float> h_a(n);

    //random number generator
    std::random_device rd;
    std::mt19937 generator(rd());
    std::uniform_real_distribution<float> dist(-1,1);

    for(size_t i=0; i<n; i++)
    {
        h_a[i] = dist(generator);
    }

    //allocate device memory
    thrust::device_vector<float> d_a(n);

    //copy host vector to device
    d_a = h_a;

    //Kernel call with timing
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    //reduce kernel
    sum = thrust::reduce(d_a.begin(), d_a.end());
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    //reduce_golden
    reduce_golden(&h_a[0], &sum_golden, n);

    //assert
    assert(abs(sum_golden - sum)<0.05);

    //prints
    //printf("sum_golden : %f\n", sum_golden);
    //printf("sum_gpu : %f\n", sum);
    printf("%f\n", sum);

    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    printf("%f\n", ms);
    //printf("Time Taken in Milliseconds : %f",ms);


    return 0;
}
