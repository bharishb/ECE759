#include "hip/hip_runtime.h"
#define HIPCUB_STDERR // print CUDA runtime errors to console
#include <stdio.h>
#include <cub/util_allocator.cuh>
#include <cub/device/device_reduce.cuh>
#include "cub/util_debug.cuh"
#include <iostream>
#include <random>
#include <cassert>
using namespace std;
using namespace hipcub;
CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory

void reduce_golden(const float* X, float* sum, unsigned int n){
    *sum = 0.0;
    for(unsigned int i=0; i<n; i++){
        *sum+=X[i];
    }
}

int main(int argc, char** argv){
	
    unsigned int n = atoi(argv[1]);
    float gpu_sum;
    float sum_golden;

    //allocate host memory
    float* h_in;
    h_in = new float[n];

    //device memory pointer
    float* d_in;

    //random number generator
    std::random_device rd;
    std::mt19937 generator(rd());
    std::uniform_real_distribution<float> dist(-1,1);

    for(size_t i=0; i<n; i++)
    {
        h_in[i] = dist(generator);
    }

    //allocate device memory
    g_allocator.DeviceAllocate((void**)& d_in, sizeof(float) * n);

    //copy host vector to device
    hipMemcpy(d_in, h_in, sizeof(float) * n, hipMemcpyHostToDevice);

    float* d_sum = NULL;
    g_allocator.DeviceAllocate((void**)& d_sum, sizeof(float) * 1);

    // Request and allocate temporary storage
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_sum, n);
    g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes);

    //Kernel call with timing
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    //reduce kernel
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_sum, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipMemcpy(&gpu_sum, d_sum, sizeof(float) * 1, hipMemcpyDeviceToHost);

    //reduce_golden
    reduce_golden(&h_in[0], &sum_golden, n);

    //assert
    assert(abs(sum_golden - gpu_sum)<0.05);

    //prints
    //printf("sum_golden : %f\n", sum_golden);
    //printf("sum_gpu : %f\n", gpu_sum);
    printf("%f\n", gpu_sum);

    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    printf("%f\n", ms);
    //printf("Time Taken in Milliseconds : %f",ms);

    //free up memory
    g_allocator.DeviceFree(d_in);
    g_allocator.DeviceFree(d_sum);
    g_allocator.DeviceFree(d_temp_storage);

    return 0;
}
