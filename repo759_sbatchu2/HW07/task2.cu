#include "count.cuh"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <iostream>
#include <random>
#include <cassert>
#include <thrust/copy.h>
using namespace std;

int main(int argc, char** argv){
	
    unsigned int n = atoi(argv[1]);

    //allocate host memory
    thrust::host_vector<int> h_a(n);
    thrust::host_vector<int> h_values(n);
    thrust::host_vector<int> h_counts(n);


    //allocate device memory
    thrust::device_vector<int> d_a(n);
    thrust::device_vector<int> values(n);
    thrust::device_vector<int> counts(n);

    //random number generator
    std::random_device rd;
    std::mt19937 generator(rd());
    std::uniform_int_distribution<int> dist(0, 500);

    for(size_t i=0; i<n; i++)
    {
        h_a[i] = dist(generator);
    }

    //copy host vector to device
    thrust::copy(h_a.begin(), h_a.end(), d_a.begin());

    //Kernel call with timing
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    count(d_a, values, counts);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
  
    //copy device vector to host
    thrust::copy(values.begin(), values.end(), h_values.begin());
    thrust::copy(counts.begin(), counts.end(), h_counts.begin());

   //printf("values size : %d\n", values.size());
   //last element of values array
   printf("%d\n", h_values[values.size()-1]);
   //printf("%d\n", values[values.size()-1]);

   //last element of counts array'
   printf("%d\n", h_counts[values.size()-1]);
   //printf("%d\n", counts[values.size()-1]);

   /*for(int i=0; i<h_a.size(); i++)
    {
        printf("h_a values %d : %d\n", i, h_a[i]);
    }
   for(int i=0; i<values.size(); i++)
    {
        printf("values %d : %d\n", i, h_values[i]);
    }

    for(int i=0; i<values.size(); i++)
    {
        printf("counts %d : %d\n", i, h_counts[i]);
    }*/
    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    printf("%f\n", ms);
    //printf("Time Taken in Milliseconds : %f",ms);


    return 0;
}
